
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <functional>
#include <hiprand/hiprand_kernel.h>

#define threadsPerBlock 256
typedef double(*test_func_t)(double*, int, double);

__device__ double rastrigin_cuda(double *input, int size, double val) {
	if (size == 0 && input == NULL) {
		return val * val - 10.0 * cos(2.0 * M_PI * val);
	}
	double first_term = 10 * static_cast<double>(size);
	double second_term = 0.0;
	for (int i = 0; i < size; ++i) {
		second_term += (input[i] * input[i]);
		second_term -= 10.0 * cos(2.0 * M_PI * input[i]);
	}
	return first_term + second_term;
}

__device__ double ackley_cuda(double *input, int size, double val) {
	if (size == 0 && input == NULL) {
		return -20.0 * exp(-0.2 * sqrt(0.5 * val * val))
			-exp(0.5 * cos(2.0 * M_PI * val));
	}
	double square_term = 0.0;
	double cosine_term = 0.0;
	for (int i = 0; i < size; ++i) {
		square_term += input[i] * input[i];
		cosine_term += cos(2.0 * M_PI * input[i]);
	}
	double first_term = -20.0 * exp(-0.2 * sqrt(0.5 * square_term));
	double second_term = -exp(cosine_term / double(size)) + exp(1.0) + 20.0;
	return first_term + second_term;
}

__global__ void sa_kernel(double *dev_solution, int size, double lo, double hi, 
		double sigma, int choice) {
	hiprandState state;
	test_func_t func;
	switch (choice) {
		case(1):
			func = rastrigin_cuda;
			break;
		case(2):
			func = ackley_cuda;
			break;
	}
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > size) return;
	hiprand_init(idx, 0, 0, &state);
	double iter = 0.0;
	double temperature = 1.0;
	double sol_idx = dev_solution[idx];
	while(temperature >= 1e-6) {
			double original_sol = sol_idx;
			double diff = -func(NULL, 0, sol_idx);
			sol_idx += hiprand_normal_double(&state) * sigma;
			diff += func(NULL, 0, sol_idx);
			if (diff > 0) {
				double alpha = hiprand_uniform_double(&state);
				double prob = exp(-diff / temperature);
				if (alpha > prob) {
					sol_idx = original_sol;
				}
			}
		temperature = 1.0 / (1.0+2.5*iter);
		iter += 1.0;
	}
	dev_solution[idx] = sol_idx;
}

void simulate_annealing_cuda(double *solution, int size, double lo,
		double hi, double sigma, float *msec, int choice) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    int numBlock = size / threadsPerBlock + 1;
	double *dev_solution;
	hipMalloc(&dev_solution, sizeof(double)*size);
	hipMemcpy(dev_solution, solution, sizeof(double)*size, hipMemcpyHostToDevice);
	hipEventRecord(start);
    sa_kernel<<<numBlock, threadsPerBlock>>>(dev_solution, size, lo, hi, sigma, choice);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(msec, start, stop);
	hipMemcpy(solution, dev_solution, sizeof(double)*size, hipMemcpyDeviceToHost);
	hipFree(&dev_solution);
}

void printCudaInfo()
{
    // for fun, just print out some stats on the machine
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<double>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
