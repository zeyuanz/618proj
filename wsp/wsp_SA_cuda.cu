
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <functional>
#include <hiprand/hiprand_kernel.h>

#define threadsPerBlock 256

typedef struct path_struct_t
{
	double cost; // path cost.
	int *path;   // best order of city visits
} path_t;

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "CUDA Error: %s at %s:%d\n",
				hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
#else
#define cudaCheckError(ans) ans
#endif


void printCudaInfo()
{
	// for fun, just print out some stats on the machine
	int deviceCount = 0;
	hipError_t err = hipGetDeviceCount(&deviceCount);

	printf("---------------------------------------------------------\n");
	printf("Found %d CUDA devices\n", deviceCount);

	for (int i=0; i<deviceCount; i++)
	{
		hipDeviceProp_t deviceProps;
		hipGetDeviceProperties(&deviceProps, i);
		printf("Device %d: %s\n", i, deviceProps.name);
		printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
		printf("   Global mem: %.0f MB\n",
				static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
		printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
	}
	printf("---------------------------------------------------------\n"); 
}



__device__ void set_dist_cu(double *dist, int n_cities, int i, int j, double value)
{
	int offset = i * n_cities + j;
	dist[offset] = value;
	return;
}

__device__ double get_dist_cu(double *dist, int n_cities, int i, int j)
{
	int offset = i * n_cities + j;
	return dist[offset];
}

__device__ void init_path_cu(path_t *cost_path, int n_cities, double *dist, hiprandState_t *state, int first)
{
	cost_path->cost = 0.0;
	// initialize path in 0->1->2->3 ... ->n
	for (int i = 0; i < n_cities; i++)
	{
		int city = i;
		cost_path->path[i] = city;
	}

	int temp = cost_path->path[first];
	cost_path->path[first] = cost_path->path[0];
	cost_path->path[0] = temp;
	// create a random permutation of the path
	for (int i = n_cities - 1; i >= 1; --i)
	{
		int j = hiprand(state) % (i) + 1;
		int temp = cost_path->path[i];
		cost_path->path[i] = cost_path->path[j];
		cost_path->path[j] = temp;
	}

	// compute the cost after permutation
	for (int i = 0; i < n_cities - 1; ++i)
	{
		cost_path->cost += get_dist_cu(dist, n_cities, cost_path->path[i], cost_path->path[i + 1]);
	}
}

__device__ double edge_dist_cu(double *dist, int n_cities, int *cost_path, int *rand_position)
{
	double cost = 0;
	// if the position is not the start
	if (*rand_position != 0)
	{
		cost += get_dist_cu(dist, n_cities, cost_path[*rand_position - 1], cost_path[*rand_position]);
	}
	// if the position is not the end
	if (*rand_position != n_cities - 1)
	{
		cost += get_dist_cu(dist, n_cities, cost_path[*rand_position], cost_path[*rand_position + 1]);
	}
	return cost;
}

__device__ void swap_city_cu(int *cost_path_path, int *rand_position_1, int *rand_position_2)
{
	int tmp = cost_path_path[*rand_position_1];
	cost_path_path[*rand_position_1] = cost_path_path[*rand_position_2];
	cost_path_path[*rand_position_2] = tmp;
}

__device__ double random_swap_city_cost_cu(path_t *cost_path, int n_cities, double *dist, int *rand_position_1, int *rand_position_2, hiprandState_t *state)
{
	double cost = cost_path->cost;
	// randomly select to cities. Make sure two cities are different.
	// also, because of search space decomposition, the first city cannot be choosen.
	*rand_position_1 = (hiprand(state) % (n_cities - 1)) + 1;
	*rand_position_2 = (hiprand(state) % (n_cities - 1)) + 1;
	while (*rand_position_1 == *rand_position_2)
	{
		*rand_position_1 = hiprand(state) % (n_cities - 1) + 1;
	}
	// minus the cost when taking out two cities from path
	cost -= edge_dist_cu(dist, n_cities, cost_path->path, rand_position_1);
	cost -= edge_dist_cu(dist, n_cities, cost_path->path, rand_position_2);
	// swap the city
	swap_city_cu(cost_path->path, rand_position_1, rand_position_2);
	// add the cost when adding two cities to the path
	cost += edge_dist_cu(dist, n_cities, cost_path->path, rand_position_1);
	cost += edge_dist_cu(dist, n_cities, cost_path->path, rand_position_2);
	return cost;
}

__global__ void wsp_sa_kernel(double *dev_all_cost, int *dev_all_path, double* dist, int n_cities)
{
	__shared__ double sdata[threadsPerBlock];
	__shared__ int idata[threadsPerBlock];
	int tid = threadIdx.x;
	idata[tid] = tid;
	hiprandState state;
	double temperature = 20.0;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(idx, 0, 0, &state);
	if (idx >= n_cities) return;
	path_t *cost_path = (path_t *)malloc(sizeof(path_t));
	cost_path->cost = 0.0;
	cost_path->path = new int[n_cities];
	init_path_cu(cost_path, n_cities, dist, &state, idx);
	int *rand_position_1 = new int(1);
	int *rand_position_2 = new int(2);
	int cnt = 0;
	while(cnt < 2000)
	{
		double original_cost = cost_path->cost;
		double new_cost = random_swap_city_cost_cu(cost_path, n_cities, dist, rand_position_1, rand_position_2, &state);
		// if new cost is smaller, accept
		if (new_cost < original_cost)
		{
			cost_path->cost = new_cost;
			cnt = 0;
		}
		else
		{
			// if new cost is bigger, accept with probability
			double diff = static_cast<double>(original_cost - new_cost);
			double prob;
			if (temperature < 1e-12)
			{
				prob = 0.0;
			}
			else
			{
				prob = exp(diff / temperature);
			}
			// obtain a random number in (0,1) to decision
			double rand_number = hiprand_uniform_double(&state);
			if (rand_number < prob)
			{
				cost_path->cost = new_cost;
				cnt = 0;
			}
			else
			{
				// if not accepted, recover the state
				swap_city_cu(cost_path->path, rand_position_1, rand_position_2);
				cnt++;
			}
		}
		// annealing step (i.e. reduce temperature)
		temperature *= 0.999999;
	}
	sdata[tid] = cost_path->cost;
	__syncthreads();

	// reduction
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		int idx1 = idx;
		int idx2 = idx1 + s;
		if (tid < s && idx1 < n_cities && idx2 < n_cities) {
			double s1 = sdata[tid];
			double s2 = sdata[tid + s];
			if(s1 > s2)
			{
				sdata[tid] = sdata[tid + s];
				idata[tid] = idata[tid + s];
			}
		}
		__syncthreads();
	}
	// now sdata[0] is the min cost, this cost is from thread minId
	if(tid == idata[0])
	{
		dev_all_cost[blockIdx.x] = sdata[0];
		memcpy(dev_all_path + blockIdx.x * n_cities, cost_path->path, sizeof(int)*n_cities);
	}
}

void wsp_simulate_annealing_cuda(path_t *solution, int n_cities, double *dist, float *msec) 
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int blocks = n_cities / threadsPerBlock + 1;
	int *dev_all_path;
	double *dev_all_cost;
	int *host_all_path = new int[n_cities * blocks];
	double *host_all_cost = new double[blocks];
	double *dist_cu;
	hipMalloc(&dev_all_path, sizeof(int)*n_cities*blocks);
	hipMalloc(&dev_all_cost, sizeof(double)*blocks);
	hipMalloc(&dist_cu, sizeof(double)*n_cities*n_cities);
	hipMemcpy(dist_cu, dist, sizeof(double)*n_cities*n_cities, hipMemcpyHostToDevice);
	hipEventRecord(start);
	wsp_sa_kernel<<<blocks, threadsPerBlock>>>(dev_all_cost, dev_all_path, dist_cu, n_cities);
	hipDeviceSynchronize();
	hipMemcpy(host_all_path, dev_all_path, sizeof(int)*n_cities*blocks, hipMemcpyDeviceToHost);
	hipMemcpy(host_all_cost, dev_all_cost, sizeof(double)*blocks, hipMemcpyDeviceToHost);
	int index = 0;
	double min = host_all_cost[0];
	for(int i = 0; i < blocks; i++) 
	{
		double local = host_all_cost[i];
		if(local < min)
		{
			index = i;
			min = local;
		}
	}
	solution->cost = host_all_cost[index];
	memcpy(solution->path, host_all_path + index * n_cities, sizeof(int)*n_cities);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(msec, start, stop);
	delete[] host_all_path;
	hipFree(&dev_all_path);
	hipFree(&dist_cu);
}
